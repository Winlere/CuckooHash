#include "hash.cuh"
#include "time.hpp"
#include "helper.cuh"
#include <iostream>
#include <vector>
const int BLOCKSIZE = 512;
int maxRetry = 50;

int main()
{
    TIME_INIT;
    const uint32_t tableSize = 1 << 25;
    const uint32_t testMaxSize = 1 << 24;
    const uint32_t seed1 = 114514;
    const uint32_t seed2 = 191981;
    const int range = 1 << 30;
    HashFunc f1, f2;
    hashTableEntry *d_hashTable = nullptr;
    initHashTable(&d_hashTable, tableSize);

    int *d_keys = nullptr;
    hipMalloc((void **)&d_keys, sizeof(int) * tableSize);
    hipMemset(d_keys, 0xff, sizeof(int) * tableSize);
    generateRandomKeys<<<(testMaxSize + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_keys, tableSize, range);
    hipDeviceSynchronize();

    int *d_retvals = nullptr;
    hipMalloc((void **)&d_retvals, sizeof(int) * tableSize);
    int testTableSize = 1.4 * testMaxSize;
    for (int maxMove = 1; maxMove <= 125; ++maxMove)
    {
        f1.seed = 115838;
        f1.tableSize = testTableSize;
        f2.seed = 193305;
        f2.tableSize = testTableSize;
    retry_entry:
        int testSize = 1 << 24;
        reuseHashTable(d_hashTable, testTableSize);
        TIME_START;
        insertItemBatch<<<(testSize + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_hashTable, d_keys, d_retvals, testTableSize, testSize, f1, f2, maxMove);
        hipDeviceSynchronize();
        TIME_END;
        bool valid = isArrayAllEqualToValue(d_retvals, testSize, 0);
        if (!valid)
        {
            ++maxRetry;
            f1 = {seed1 + maxRetry, (unsigned) testTableSize};
            f2 = {seed2 + maxRetry, (unsigned) testTableSize};
            std::cout << "retrying..." << std::endl;
            goto retry_entry;
        }
        std::cout << "maxMove,elapsed_μs,valid | " << maxMove << "," << elapsed_μs << "," << valid << std::endl;
    }
    hipFree(d_keys);
    hipFree(d_retvals);
    hipFree(d_hashTable);
    return 0;
}