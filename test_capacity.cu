#include "hash.cuh"
#include "time.hpp"
#include "helper.cuh"
#include <iostream>
#include <vector>

int main()
{
    TIME_INIT;
    const uint32_t maxTableSize = 1 << 25;
    const uint32_t testMaxSize = 1 << 24;
    const uint32_t seed1 = 114514;
    const uint32_t seed2 = 191981;
    const int range = 1 << 30;
    int retries = 0;
    hashTableEntry *d_hashTable = nullptr;
    initHashTable(&d_hashTable, maxTableSize);

    int *d_keys = nullptr;
    hipMalloc((void **)&d_keys, sizeof(int) * maxTableSize);
    hipMemset(d_keys, 0xff, sizeof(int) * maxTableSize);
    generateRandomKeys<<<(maxTableSize + 255) / 256, 256>>>(d_keys, maxTableSize, range);
    hipDeviceSynchronize();
    HashFunc f1,f2;

    int *d_retvals = nullptr;
    hipMalloc(&d_retvals, sizeof(int) * maxTableSize);
    {

        uint32_t testSize = testMaxSize;
        std::vector<int> tableSizeProportions = {110, 120, 130, 140, 150, 160, 170, 180, 190, 200, 101, 102, 105};
        for (int proportion : tableSizeProportions)
        {
            uint32_t tableSize = testMaxSize * proportion / 100;
            retries = 0;
            f1 = {seed1 + retries, tableSize}, f2 = {seed2 + retries, tableSize};
            reuseHashTable(d_hashTable, tableSize);
            TIME_START;
            insertItemBatch<<<(testSize + 256 - 1) / 256, 256>>>(d_hashTable, d_keys, d_retvals, tableSize, testSize, f1, f2);
            hipDeviceSynchronize();
            TIME_END;
            // // print the first 100000 values of d_retvals
            // int *h_retvals = new int[testMaxSize];
            // hipMemcpy(h_retvals, d_retvals, sizeof(int) * testMaxSize, hipMemcpyDeviceToHost);
            // std::cout << "d_retvals[0:9] = ";
            // for (int i = 0; i < 128; i++)
            // {
            //     std::cout << h_retvals[i] << " ";
            // }
            // std::cout << std::endl;
            // delete [] h_retvals;
            bool valid = isArrayAllEqualToValue(d_retvals, (int)testMaxSize, 0);
            // bool valid = 1;
            hipDeviceSynchronize();

            std::cout << "tableSize,elapsed_μs,valid | " << tableSize << "," << elapsed_μs << "," << valid << std::endl;            
        }
    }  
    hipFree(d_keys);
    hipFree(d_retvals); 
    hipFree(d_hashTable);
    return 0;  
}
