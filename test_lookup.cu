#include "hip/hip_runtime.h"
#include "hash.cuh"
#include "time.hpp"
#include "helper.cuh"
#include <iostream>
#include <vector>
#include <unordered_set>
#include <algorithm>
#include <omp.h>
#include <random>

int retry_times = 900;
int main()
{
    TIME_INIT;
#ifdef TRIHASH
    const uint32_t tableSize = 1 << 25;
#else
    const uint32_t tableSize = 1 << 26;
#endif
    const uint32_t testMaxSize = 1 << 24;
    const uint32_t seed1 = 114514;
    const uint32_t seed2 = 191981;
    const int range = 1 << 30;
    hashTableEntry *d_hashTable = nullptr;
    initHashTable(&d_hashTable, tableSize);

    int *d_keys = nullptr;
    hipMalloc((void **)&d_keys, sizeof(int) * tableSize);
    hipMemset(d_keys, 0xff, sizeof(int) * tableSize);
    generateRandomKeys<<<(testMaxSize + 255) / 256, 256>>>(d_keys, tableSize, range, seed1);
    hipDeviceSynchronize();

    HashFunc f1{seed1 + retry_times, tableSize}, f2{seed2 + retry_times, tableSize};

    int *d_retvals = nullptr;
    hipMalloc((void **)&d_retvals, sizeof(int) * tableSize);
    {
        int testSize;
    retry:
        f1 = {seed1 + retry_times, tableSize};
        f2 = {seed2 + retry_times, tableSize};
        testSize = testMaxSize;
        reuseHashTable(d_hashTable, tableSize);
        TIME_START;
        insertItemBatch<<<(testSize + 255) / 256, 256>>>(d_hashTable, d_keys, d_retvals, tableSize, testSize, f1, f2, MAX_MOVE_TIME);
        hipDeviceSynchronize();
        TIME_END;
        bool valid = isArrayAllEqualToValue(d_retvals, testSize, 0);
        if (valid){
            // std::cout << "construction sucessfull" << std::endl;
        }else
        {
            // std::cout << "failed. reconstructing..." << std::endl;
            ++retry_times;
            goto retry;
        }
        // report sucess hash parameters
        // std::cout << "report sucess hash parameters" << std::endl;
        // std::cout << "f1.seed = " << f1.seed << " f1.tableSize = " << f1.tableSize << std::endl;
        // std::cout << "f2.seed = " << f2.seed << " f2.tableSize = " << f2.tableSize << std::endl;
    }

    // random shuffle the old keys. to ensure uniformity
    std::vector<int> oldKeys(testMaxSize);
    hipMemcpy(oldKeys.data(), d_keys, sizeof(int) * testMaxSize, hipMemcpyDeviceToHost);
    std::random_shuffle(oldKeys.begin(), oldKeys.end());
    std::unordered_set<int> oldKeysSet(oldKeys.begin(), oldKeys.end());
    hipMemcpy(d_keys, oldKeys.data(), sizeof(int) * testMaxSize, hipMemcpyHostToDevice);

    // prepare randomkeys
    int *d_queries = nullptr;
    hipMalloc((void **)&d_queries, sizeof(int) * testMaxSize);
    std::vector<int> h_newRandomKeys(testMaxSize);
#pragma omp parallel for shared(testMaxSize, range, h_newRandomKeys,oldKeysSet) schedule(dynamic)
    for (uint32_t i = 0; i < testMaxSize; ++i)
    {
        std::mt19937 rng(i);
        std::uniform_int_distribution<int> uni(0, range);
        int key = uni(rng);
        while (oldKeysSet.find(key) != oldKeysSet.end())
        {
            key = uni(rng);
        }
        h_newRandomKeys[i] = key;
    }
    int *d_newRandomKeys = nullptr;
    hipMalloc((void **)&d_newRandomKeys, sizeof(int) * testMaxSize);
    hipMemcpy(d_newRandomKeys, h_newRandomKeys.data(), sizeof(int) * testMaxSize, hipMemcpyHostToDevice);
    for (int i = 0; i <= 10; ++i)
    {
        int existingTestSize = std::max(testMaxSize * i / 10, 1u);
        int randomTestSize = testMaxSize - existingTestSize;
        // copy the existing keys to the new keys
        hipMemcpy(d_queries, d_keys, sizeof(int) * existingTestSize, hipMemcpyDeviceToDevice);
        hipMemcpy(d_queries + existingTestSize, d_newRandomKeys, sizeof(int) * randomTestSize, hipMemcpyDeviceToDevice);
        // do lookups
        TIME_START;
        lookupItemBatch<<<(testMaxSize + 512 - 1) / 512, 512>>>(d_hashTable, d_queries, d_retvals, tableSize, testMaxSize, f1, f2);
        hipDeviceSynchronize();
        TIME_END;
        // validate the return values
        // bool valid = isArrayAllNotEqualToValue(d_retvals, existingTestSize, NOT_A_INDEX) && isArrayAllEqualToValue(d_retvals + existingTestSize, randomTestSize, NOT_A_INDEX);
        bool valid1 = isArrayAllNotEqualToValue(d_retvals, existingTestSize, NOT_A_INDEX);
        bool valid2 = isArrayAllEqualToValue(d_retvals + existingTestSize, randomTestSize, NOT_A_INDEX);
        bool valid = valid1 && valid2;
        // std::cout << "valid1,2,#=" << valid1 << ',' << valid2 << ',' << valid << std::endl;
        std::cout << "percentage,elapsed_μs,valid | " << i << ',' << elapsed_μs << "," << valid << std::endl;
    }

    hipFree(d_keys);
    hipFree(d_retvals);
    hipFree(d_queries);
    hipFree(d_newRandomKeys);
    hipFree(d_hashTable);
    return 0;
}