#include "hash.cuh"
#include "time.hpp"
#include "helper.cuh"
#include <iostream>
#include <vector>

int main()
{
    TIME_INIT;
    const uint32_t tableSize = 1 << 25;
    const uint32_t testMaxSize = 1 << 24;
    const uint32_t seed1 = 114514;
    const uint32_t seed2 = 191981;
    hashTableEntry *d_hashTable = nullptr;
    initHashTable(&d_hashTable, tableSize);

    int *d_keys = nullptr;
    hipMalloc((void **)&d_keys, sizeof(int) * tableSize);

    int *d_retvals = nullptr;
    hipMalloc((void **)&d_retvals, sizeof(int) * testMaxSize);

    for (uint32_t s = 11; s <= 24; ++s)
    {
        int testSize = 1 << s;
        HashFunc f1{seed1 * s, tableSize}, f2{seed2 * s, tableSize};
        TIME_START;
        insertItemBatch<<<(testSize + 255) / 256, 256>>>(d_hashTable, d_keys, d_retvals, tableSize, testSize, f1, f2);
        hipDeviceSynchronize();
        TIME_END;
        bool valid = isArrayAllEqualToValue(d_retvals, testSize, 0);
        std::cout << "testsize,elapsed_ms,valid | " << testSize << " " << elapsed_ms << " " << valid << std::endl;
    }
    return 0;
}