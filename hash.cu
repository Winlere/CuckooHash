#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <unordered_set>
#include <cstring>
#include <map>
#include "hash.cuh"
#define MAX_MOVE_TIME 100
#define NOT_A_KEY -1 // if change this, please change the memset value as well.
#define NOT_A_INDEX -1

class hashTableEntry
{
public:
    int key;
    // int value; no value is needed
};

int initHashTable(hashTableEntry **d_table, int tableSize)
{
    hipMalloc(d_table, sizeof(hashTableEntry) * tableSize);
    hipMemset(*d_table, 0xff, sizeof(hashTableEntry) * tableSize);
    return 0;
}

int reuseHashTable(hashTableEntry **d_table, int tableSize){
    hipMemset(d_table, 0xff, sizeof(hashTableEntry) * tableSize);
    return 0;
}



__device__ inline void insertItem(hashTableEntry *d_table, int original_key, HashFunc f1, HashFunc f2, int *retval)
{
    *retval = 0;
    int move_time = 0;
    int h1 = f1(original_key);
    int h2 = f2(original_key);
    int evicteeIndex = h1;
    if (d_table[h1].key == original_key || d_table[h2].key == original_key)
        return; // Duplicate key
    // Try to place original_key in the slot
    int current_key = original_key;
    int k1 = atomicExch(&d_table[h1].key, current_key);
    if (k1 == NOT_A_KEY)
        return;

    current_key = k1; // Now we work with the evicted key

    do
    { // This block tries to place 'current_key' in the alternative slot
        h1 = f1(current_key);
        h2 = f2(current_key);
        int alternativeIndex = evicteeIndex == h1 ? h2 : h1;
        k1 = atomicExch(&d_table[alternativeIndex].key, current_key);
        if (k1 == NOT_A_KEY)
            return;

        current_key = k1; // Update the current_key with the newly evicted key
        evicteeIndex = alternativeIndex;
        ++move_time;
    } while (move_time < MAX_MOVE_TIME);

    *retval = 1; // Indicate failure after MAX_MOVE_TIME attempts
}

__device__ inline void lookupItem(hashTableEntry *d_table, int key, HashFunc f1, HashFunc f2, int *retval)
{
    *retval = NOT_A_INDEX;
    int h1 = f1(key);
    if (d_table[h1].key == key)
    {
        *retval = d_table[h1].key;
        return;
    }
    int h2 = f2(key);
    if (d_table[h2].key == key)
    {
        *retval = d_table[h2].key;
        return;
    }
    *retval = NOT_A_INDEX;
}

__global__ void insertItemBatch(hashTableEntry *d_table, int *d_keys, int *d_retvals, int tableSize, int batchSize, HashFunc f1, HashFunc f2)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= batchSize)
        return;
    int key = d_keys[tid];
    int *retval = d_retvals + tid;
    insertItem(d_table, key, f1, f2, retval);
}

__global__ void lookupItemBatch(hashTableEntry *d_table, int *d_keys, int *d_retvals, int tableSize, int batchSize, HashFunc f1, HashFunc f2)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= batchSize)
        return;
    int key = d_keys[tid];
    int *retval = d_retvals + tid;
    lookupItem(d_table, key, f1, f2, retval);
}

bool validation(int tableSize_, int test_)
{
    std::cout << "[Sanity Check] start sanity check"
              << "tableSize=" << tableSize_ << " test=" << test_ << std::endl;
    std::mt19937_64 rng(123);
    std::uniform_int_distribution<int> dist(0, 1e9);
    uint32_t tableSize = tableSize_;
    uint32_t test = test_;
    std::vector<int> keys;
    std::unordered_set<int> keySet;
    for (uint32_t i = 0; i < test; ++i)
    {
        int key = dist(rng);
        if (keySet.find(key) == keySet.end())
        {
            keySet.insert(key);
            keys.push_back(key);
        }
    }
    hashTableEntry *d_table;
    hipMalloc(&d_table, sizeof(hashTableEntry) * tableSize);
    const int MAX_RETRY = 30;
    int retries = MAX_RETRY;
    HashFunc f1, f2;
    std::map<std::pair<int, int>, int> dup;
    uint32_t a1, a2;
retry:
    hipMemset(d_table, 0xff, sizeof(hashTableEntry) * tableSize);
    a1 = dist(rng), a2 = dist(rng);
    f1 = HashFunc{a1, tableSize};
    f2 = HashFunc{a2, tableSize};
    dup.clear();
    for (int key : keys)
    {
        int h1 = f1(key);
        int h2 = f2(key);
        dup[{h1, h2}]++;
        // std:: cout << key << " " << h1 << " " << h2 << std::endl;
        if (dup[{h1, h2}] == 3)
        {
            // LOG_DEBUG(("inherent collision detected. h1 = %d h2 = %d",h1,h2));
            for (auto key2 : keys)
            {
                int h1_ = f1(key2); // hash(key2,a1,b1,tableSize);
                int h2_ = f2(key2); // hash(key2,a2,b2,tableSize);
                if (h1 == h1_ && h2 == h2_)
                    std::cout << "[Sanity Check] key,h1,h2 = " << key2 << " " << h1 << " " << h2 << std::endl;
            }
            if (retries--)
            {
                std::cout << "[Sanity Check] retrying..." << std::endl;
                goto retry;
            }
            else
            {
                std::cout << "[Sanity Check] failed after " + std::to_string(MAX_RETRY) + " retries. aborting..." << std::endl;
                hipFree(d_table);
                return false;
            }
        }
    }
    std::cout << "[Sanity Check] no inherent hash collision detected." << std::endl;
    // insert test
    for (int key : keys)
    {
        int *d_retval;
        int *d_key;
        hipMalloc(&d_key, sizeof(int));
        hipMemcpy(d_key, &key, sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&d_retval, sizeof(int));
        insertItemBatch<<<1, 1>>>(d_table, d_key, d_retval, tableSize, 1, f1, f2);
        hipDeviceSynchronize();
        int retval;
        hipMemcpy(&retval, d_retval, sizeof(int), hipMemcpyDeviceToHost);
        if (retval == 1)
        {
            std::cout << "[Sanity Check] insertion failed (or reconstruction needed)" << std::endl;

            if (retries--)
            {
                std::cout << "[Sanity Check] retrying..." << std::endl;
                goto retry;
            }
            else
            {
                std::cout << "[Sanity Check] failed after " + std::to_string(MAX_RETRY) + " retries. aborting..." << std::endl;
                hipFree(d_table);
                return false;
            }
            hipFree(d_table);
        }
    }
    std::cout << "[Sanity Check] passed insert test." << std::endl;
    // lookup test
    for (int key : keys)
    {
        int *d_retval;
        int *d_key;
        hipMalloc(&d_key, sizeof(int));
        hipMemcpy(d_key, &key, sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&d_retval, sizeof(int));
        lookupItemBatch<<<1, 1>>>(d_table, d_key, d_retval, tableSize, 1, f1, f2);
        hipDeviceSynchronize();
        int retval;
        hipMemcpy(&retval, d_retval, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_retval);
        if (retval == NOT_A_INDEX)
        {
            std::cout << "[Sanity Check] lookup failed (false negative)" << std::endl;
            std::cout << "[Sanity Check] key = " << key << std::endl;
            std::cout << "[Sanity Check] h1 = " << f1(key) << std::endl;
            std::cout << "[Sanity Check] h2 = " << f2(key) << std::endl;
            std::cout << "[Sanity Check] retval = " << retval << std::endl;
            hipFree(d_table);
            return false;
        }
    }
    std::cout << "[Sanity Check] passed lookup test. no false negative" << std::endl;
    // lookup test
    for (uint32_t i = 0; i < test; ++i)
    {
        int key;
        do
        {
            key = dist(rng);
        } while (keySet.find(key) != keySet.end());
        int *d_retval;
        int *d_key;
        hipMalloc(&d_key, sizeof(int));
        hipMemcpy(d_key, &key, sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&d_retval, sizeof(int));
        lookupItemBatch<<<1, 1>>>(d_table, d_key, d_retval, tableSize, 1, f1, f2);
        hipDeviceSynchronize();
        int retval;
        hipMemcpy(&retval, d_retval, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_retval);
        if (retval != NOT_A_INDEX)
        {
            std::cout << "[Sanity Check] lookup failed (false positive)" << std::endl;
            hipFree(d_table);
            return false;
        }
    }
    std::cout << "[Sanity Check] passed lookup test. no false positive" << std::endl;
    std::cout << "[Sanity Check] passed all sanity tests!" << std::endl;
    // release resource
    hipFree(d_table);
    return true;
}

__global__ void generateRandomKeys(int *d_keys, int batchSize, int range)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= batchSize)
        return;
    HashFunc f{tid,(uint32_t)range};
    d_keys[tid] = f(tid);
}